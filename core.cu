#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <float.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include "common.h"
#include "core.h"

//#include "core_openmp.h"

extern double size;
#define cutoff 0.01  // TODO: check Piazza if this will ever change

//
// Custom classes
//

//
// get bounds in overall domain
//
void get_bounds(int nsquares_per_side, double* vertbounds, double* horzbounds)
{
    vertbounds[0] = 0.0;
    horzbounds[0] = 0.0;
    for (int i = 1; i < nsquares_per_side+1; ++i){
        vertbounds[i] = size*i/nsquares_per_side;
        horzbounds[i] = size*i/nsquares_per_side;
    }
}

double get_box_width(int nsquares_per_side) {
    return size / nsquares_per_side;
}

//
// get 9 neighbors of each box (one of those being itself). if box has less neighbors, padded with -1
//
void get_box_neighbors(int nsquares, int nsquares_per_side, int* boxneighbors)
{
    for (int i = 0; i < nsquares*9; ++i) {
        boxneighbors[i] = -1;
    }
    for (int i = 0; i < nsquares; ++i){
        int row = floor(i/nsquares_per_side);
        int col = i%nsquares_per_side;
        for (int j = -1; j <= 1; ++j){
            for (int k = -1; k <= 1; ++k){
                if (
                    ((row+j) > -1) &&
                    ((row+j) < nsquares_per_side) &&
                    ((col+k) > -1) &&
                    ((col+k) < nsquares_per_side)
                ) {
                    int neighbor_i = (row+j)*nsquares_per_side + (col+k);
                    boxneighbors[i*9 + (j+1)*3 + k+1] = neighbor_i;
                }
            }
        }
    }
}

//
// classify particles into their respective bins and store the particle numbers in bin_contents
// count how many particles are in each bin and store nparticles_per_bin
//
void bin_particles(particle_t* particledata, int* nparticles_per_bin, int* bin_contents, double* vertbounds, double* horzbounds, int nsquares_per_side, int n)
{
    for (int part = 0; part < n; ++part){
        int flag = 0;
        for (int i = 0; i < nsquares_per_side; ++i){
            for (int j = 0; j < nsquares_per_side; ++j){
                if ((particledata[part].x >= vertbounds[i]) && (particledata[part].x < vertbounds[i+1]) && (particledata[part].y >= horzbounds[j]) && (particledata[part].y < horzbounds[j+1])){
                    int bin = j*nsquares_per_side + i;
                    bin_contents[bin*n + nparticles_per_bin[bin]] = part;
                    nparticles_per_bin[bin] += 1;
                    flag = 1;
                    break;
                }
            }
        }
        if (flag == 0){
            printf("WARNING: PARTICLE %i AT x=%f y=%f WAS NOT ASSIGNED TO ANY BOX\n", part, particledata[part].x, particledata[part].y);
        }
    }
}

int get_max_nsquares_per_side() {
    return floor(size / cutoff);
}
